#include "cuda_timer.cuh"

int timer_init(cuda_timer *t) {
      if (hipEventCreate(&t->start) != hipSuccess)
            return -1;
      if (hipEventCreate(&t->stop) != hipSuccess)
            return -1;
      return 0;
}

void timer_start(cuda_timer *t, hipStream_t stream) {
      hipEventRecord(t->start, stream);
}

double timer_stop(cuda_timer *t, hipStream_t stream) {
      hipEventRecord(t->stop, stream);
      hipEventSynchronize(t->stop);
      float ms = 0.0f;
      hipEventElapsedTime(&ms, t->start, t->stop);
      return (double)ms;
}

void timer_destroy(cuda_timer *t) {
      hipEventDestroy(t->start);
      hipEventDestroy(t->stop);
}
